
#include <hip/hip_runtime.h>
void __global__ fftshiftc2d(float2 *f, int det, int ntheta) {
  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = blockDim.y * blockIdx.y + threadIdx.y;
  int tz = blockDim.z * blockIdx.z + threadIdx.z;
  if (tx >= det || ty >= det || tz >= ntheta)
    return;
  int g = (1 - 2 * ((tx + 1) % 2))*(1 - 2 * ((ty + 1) % 2));
  int f_ind = tx + ty * det  + tz * det * det;
  f[f_ind].x *= g;
  f[f_ind].y *= g;
}

void __global__ fftshiftc3d(float2 *f, int n0, int n1, int n2) {
  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = blockDim.y * blockIdx.y + threadIdx.y;
  int tz = blockDim.z * blockIdx.z + threadIdx.z;
  if (tx >= n0 || ty >= n1 || tz >= n2)
    return;
  int g = (1 - 2 * ((tx + 1) % 2)) * (1 - 2 * ((ty + 1) % 2))* (1 - 2 * ((tz + 1) % 2));
  f[tx + ty * n0 + tz * n0 * n1].x *= g;
  f[tx + ty * n0 + tz * n0 * n1].y *= g; 
}
